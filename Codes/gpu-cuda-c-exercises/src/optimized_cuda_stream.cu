#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../common/book.h"

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)


__global__ void kernel( int *a, int *b, int *c ) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N) {
	int idx1 = (idx + 1) % 256;
	int idx2 = (idx + 2) % 256;
	float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
	float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
	c[idx] = (as + bs) / 2;
	}
}

int main( void ) {
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice( &whichDevice );
	hipGetDeviceProperties( &prop, whichDevice );
	if (!prop.deviceOverlap) {
		printf( "Device will not handle overlaps, so no "
		"speed up from streams\n" );
		return 0;
	}
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	hipStream_t stream0, stream1;
	hipStreamCreate( &stream0 );
	hipStreamCreate( &stream1 );

	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	hipMalloc( (void**)&dev_a0,N * sizeof(int) );
	hipMalloc( (void**)&dev_b0,N * sizeof(int) );
	hipMalloc( (void**)&dev_c0,N * sizeof(int) );
	hipMalloc( (void**)&dev_a1,N * sizeof(int) );
	hipMalloc( (void**)&dev_b1,N * sizeof(int) );
	hipMalloc( (void**)&dev_c1,N * sizeof(int) );

	hipHostAlloc( (void**)&host_a,FULL_DATA_SIZE*sizeof(int),hipHostMallocDefault );
	hipHostAlloc( (void**)&host_b,FULL_DATA_SIZE*sizeof(int),hipHostMallocDefault );
	hipHostAlloc( (void**)&host_c,FULL_DATA_SIZE*sizeof(int),hipHostMallocDefault );
	for (int i=0; i<FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	// now loop over full data, in bite-sized chunks
	for (int i=0; i<FULL_DATA_SIZE; i+= N*2) {

		hipMemcpyAsync( dev_a0, host_a+i,N * sizeof(int),hipMemcpyHostToDevice,
						stream0 );
		hipMemcpyAsync( dev_a1, host_a+i+N,N * sizeof(int),hipMemcpyHostToDevice,
						stream1 );

		hipMemcpyAsync( dev_b0, host_b+i,N * sizeof(int),hipMemcpyHostToDevice,
						stream0 );
		hipMemcpyAsync( dev_b1, host_b+i+N,N * sizeof(int),hipMemcpyHostToDevice,
						stream1 );

		kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0 );
		kernel<<<N/256,256,0,stream1>>>( dev_a1, dev_b1, dev_c1 );

		hipMemcpyAsync( host_c+i, dev_c0,N * sizeof(int),hipMemcpyDeviceToHost,
						stream0 );
		hipMemcpyAsync( host_c+i+N, dev_c1,N * sizeof(int),hipMemcpyDeviceToHost,
						stream1 );
	}

	 hipStreamSynchronize( stream0 );
	 hipStreamSynchronize( stream1 );

	 hipEventRecord( stop, 0 );
	 hipEventSynchronize( stop );
	 hipEventElapsedTime( &elapsedTime,
			 	 	 	 start, stop );
	 printf( "Time taken: %3.1f ms\n", elapsedTime );

	 hipHostFree( host_a );
	 hipHostFree( host_b );
	 hipHostFree( host_c );
	 hipFree( dev_a0 );
	 hipFree( dev_b0 );
	 hipFree( dev_c0 );
	 hipFree( dev_a1 );
	 hipFree( dev_b1 );
	 hipFree( dev_c1 );
	 hipStreamDestroy( stream0 );
	 hipStreamDestroy( stream1 );
	 return 0;
}
